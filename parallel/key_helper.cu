#include "hip/hip_runtime.h"
/******************************************************************************
 * File Name    - key_helper.cu
 * 
 * Description  - This cpp file contains the function definitions of all key 
 *                helper functions
 ******************************************************************************/

#include "key_helper.cuh"
#include "aes_parallel.cuh"
#include "omp.h"

/*******************************************************************************
* Global constants
*******************************************************************************/
// Round constant buffer
static const uint8_t round_const[10] = {0x01, 0x02, 0x04, 0x08, 0x10, 0x20, 0x40, 0x80, 0x1b, 0x36};

// Function for key expansion
void key_helper_create_round_keys(uint8_t aes_mode, uint8_t aes_key_length, const uint8_t* key, uint8_t* round_key)
{
    uint8_t i = 0;
    uint8_t num_rounds;

    if(aes_key_length == AES128_KEY_SIZE*8)
    {
        num_rounds = 10;
    }
    else
    {
        num_rounds = 14;
    }

#if (ENABLE_NAIVE | !ENABLE_OPENMP)
    // First block of round key is same as the original key
    for(; i < 16; i++)
    {
        round_key[i] = key[i];
    }

    for(; num_rounds > 0; num_rounds--)
    {
        // Generate subsequent round key blocks
        generate_round_key(round_key, i);
        i+=16;
    }
#endif

#if ENABLE_OPENMP
    #pragma omp parallel sections num_threads(4)
    {
        /* OpenMP implementation considers one column at a time which has lesser 
         * dependencies compared to row wise implementation. Each core 
         * handles 1 column.
         */
        #pragma omp section
        {
            key_helper_generate_round_key_per_core(key, round_key, 3, num_rounds);
        }

        #pragma omp section
        {
            key_helper_generate_round_key_per_core(key, round_key, 0, num_rounds);
        }
        
        #pragma omp section
        {
            key_helper_generate_round_key_per_core(key, round_key, 1, num_rounds);
        }

        #pragma omp section
        {
            key_helper_generate_round_key_per_core(key, round_key, 2, num_rounds);
        }
    }
#endif
}

#if ENABLE_OPENMP
void key_helper_generate_round_key_per_core(const uint8_t* key, uint8_t* round_key, uint8_t init_index, uint8_t num_rounds)
{
    uint8_t prev_index;
    uint8_t curr_index = init_index;

    // Copy the row corresponding to the index
    round_key[curr_index] = key[curr_index];
    round_key[curr_index + 4] = key[curr_index + 4];
    round_key[curr_index + 8] = key[curr_index + 8];

    curr_index = curr_index + 12;
    round_key[curr_index] = key[curr_index];

    for(uint8_t i = 0; i < num_rounds; i++)
    {
        #pragma omp barrier
        uint8_t temp;

        prev_index = curr_index;
        
        // Shift row step. Here instead of shifting the elements, index is shifted
        curr_index = (curr_index % 16 == 12) ? (curr_index + 3) : (curr_index - 1);

        // Substitute step. Value is fetched from the S-Box and XORed with round constant simultaneously
        temp = (curr_index % 16 == 12) ? (aes_sbox_get_val(round_key[prev_index]) ^ round_const[i]) : (aes_sbox_get_val(round_key[prev_index]));

        curr_index += 4;
        // Substituted value is XORed with the previous value in the same index
        round_key[curr_index] = round_key[curr_index - 16] ^ temp;

        curr_index +=4;
        // Previous value in the same index is XORed with the calculated previous row value
        round_key[curr_index] = round_key[curr_index - 4] ^ round_key[curr_index - 16];

        curr_index += 4;
        round_key[curr_index] = round_key[curr_index - 4] ^ round_key[curr_index - 16];

        curr_index +=4;
        round_key[curr_index] = round_key[curr_index - 4] ^ round_key[curr_index - 16];

        /* This is repeated num_rounds times. The dependency on previous rounds in 
         * AES key expansion step is completely removed.
         */
    }
}
#endif

#if (ENABLE_NAIVE | !ENABLE_OPENMP)
void generate_round_key(uint8_t* round_key, int offset)
{
    // Calculates one block of AES key
    uint8_t temp_col[4];
    uint8_t temp_byte;
    int prev_key_offset = offset - 16;

    int num_rounds_compl = offset/16;

    // Initial value - the last column of the previous block (column wise)
    temp_col[0] = round_key[prev_key_offset + 12];
    temp_col[1] = round_key[prev_key_offset + 13];
    temp_col[2] = round_key[prev_key_offset + 14];
    temp_col[3] = round_key[prev_key_offset + 15];

    // Shift column by 1
    temp_byte = temp_col[0];
    temp_col[0] = temp_col[1];
    temp_col[1] = temp_col[2];
    temp_col[2] = temp_col[3];
    temp_col[3] = temp_byte;

    // Substitute step
    temp_col[0] = aes_sbox_get_val(temp_col[0]);
    temp_col[1] = aes_sbox_get_val(temp_col[1]);
    temp_col[2] = aes_sbox_get_val(temp_col[2]);
    temp_col[3] = aes_sbox_get_val(temp_col[3]);

    // XOR the first element with round constant
    temp_col[0] ^= round_const[num_rounds_compl - 1];

    // XOR the calculated column with previous column in the same index
    round_key[offset] = round_key[prev_key_offset] ^ temp_col[0];
    round_key[offset + 1] = round_key[prev_key_offset + 1] ^ temp_col[1];
    round_key[offset + 2] = round_key[prev_key_offset + 2] ^ temp_col[2];
    round_key[offset + 3] = round_key[prev_key_offset + 3] ^ temp_col[3];

    for(int j = 4; j < 16; j+=4)
    {
        /* The subsequent columns are obtained by XOR of elements of column-1 calculated 
         * previously and the elemnts of the same column of previous block
         */
        round_key[offset + j] = round_key[prev_key_offset + j] ^ round_key[offset + j - 4];
        round_key[offset + j + 1] = round_key[prev_key_offset + j + 1] ^ round_key[offset + j - 4 + 1];
        round_key[offset + j + 2] = round_key[prev_key_offset + j + 2] ^ round_key[offset + j - 4 + 2];
        round_key[offset + j + 3] = round_key[prev_key_offset + j + 3] ^ round_key[offset + j - 4 + 3];
    }
}
#endif
