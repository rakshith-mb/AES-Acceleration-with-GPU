#include "hip/hip_runtime.h"
/******************************************************************************
 * File Name    - aes_parallel.cu
 * 
 * Description  - This cu file contains the function definitions of all aes 
 *                helper functions
 ******************************************************************************/
#include "string.h"
#include <hip/hip_runtime.h>

#include "aes_parallel.cuh"

/*******************************************************************************
* Global constants
*******************************************************************************/

static const uint8_t sbox[SBOX_LENGTH] = {
    /*          0     1     2     3     4     5     6     7     8     9     A     B     C     D     E     F */
    /* 0 */  0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76,
    /* 1 */  0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0,
    /* 2 */  0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15,
    /* 3 */  0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75,
    /* 4 */  0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84,
    /* 5 */  0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf,
    /* 6 */  0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8,
    /* 7 */  0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2,
    /* 8 */  0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73,
    /* 9 */  0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb,
    /* A */  0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79,
    /* B */  0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08,
    /* C */  0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a,
    /* D */  0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e,
    /* E */  0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf,
    /* F */  0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16 
    };

// Combined buffer to save shift row constants and matrix of mix column step
static const int8_t comb_const[32] = {0, 12, 8, 4, 0, -4, 8, 4, 0, -4, -8, 4, 0, -4, -8, -12, 2, 3, 1, 1, 1, 2, 3, 1, 1, 1, 2, 3, 3, 1, 1, 2};

// Ref - https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// Helper function to get the S-Box value
uint8_t aes_sbox_get_val(uint8_t byte_val)
{
    return sbox[byte_val];
}

// Function to initialize the AES config structure
void aes_init(aes_struct* aes_config_struct)
{
    aes_config_struct->aes_mode = AES_MODE;
    aes_config_struct->aes_key_length = AES_KEY_SIZE;

    if(aes_config_struct->aes_key_length == AES128_KEY_SIZE*8)
    {
        aes_config_struct->round_key = new uint8_t[AES128_ROUND_KEY_LENGTH];
        aes_config_struct->round_key_length = AES128_ROUND_KEY_LENGTH;
    }
    else
    {
        aes_config_struct->round_key = new uint8_t[AES256_ROUND_KEY_LENGTH];
        aes_config_struct->round_key_length = AES256_ROUND_KEY_LENGTH;
    }
}

// Function to launch the appropriate AES mode
void aes_encrypt_buffer(aes_struct* aes_config_struct)
{
    if(aes_config_struct->aes_mode == AES_ECB)
    {
        aes_encrypt_ecb(aes_config_struct);
    }
    else
    {
        aes_encrypt_ctr(aes_config_struct);
    }
}

// Function to encrypt the buffer in CTR mode
void aes_encrypt_ctr(aes_struct* aes_config_struct)
{
    // Calculate the number of blocks needed
    int block_count = (aes_config_struct->plain_text_length + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK;

    // Calculate the size required for the shared memory
    int smem_size = sizeof(uint8_t) * (SBOX_LENGTH + 2*THREADS_PER_BLOCK + AES256_ROUND_KEY_LENGTH) + sizeof(int8_t) * 32;

    uint8_t num_rounds;

    if(aes_config_struct->aes_key_length == AES128_KEY_SIZE*8)
    {
        num_rounds = 11;
    }
    else
    {
        num_rounds = 15;
    }

    uint8_t* plain_text_buf = aes_config_struct->plain_text;

    // IV is encrypted in CTR mode
    uint8_t* ctr_buf = new uint8_t[aes_config_struct->plain_text_length];

    for(int i = 0; i < aes_config_struct->plain_text_length; i++)
    {
        // Copy the IV to the buffer. IV is incremented for each block
        if((i % AES_BLK_LENGTH == 15) && (i != 15))
        {
            ctr_buf[i] = ctr_buf[i - AES_BLK_LENGTH] + 1;
        }
        else
        {
            ctr_buf[i] = aes_config_struct->counter[i % 16];
        }
    }

    // Reuse of the same code used for ECB mode
    aes_config_struct->plain_text = ctr_buf;

    uint8_t *dev_sbox_arr, *dev_round_key, *dev_plain_text, *dev_cipher_text;
    int8_t *dev_comb_arr;
    hipMalloc((void**)&dev_sbox_arr, sizeof(uint8_t) * SBOX_LENGTH);
    hipMalloc((void**)&dev_round_key, sizeof(uint8_t) * aes_config_struct->round_key_length);
    hipMalloc((void**)&dev_plain_text, sizeof(uint8_t) * aes_config_struct->plain_text_length);
    hipMalloc((void**)&dev_cipher_text, sizeof(uint8_t) * aes_config_struct->plain_text_length);
    hipMalloc((void**)&dev_comb_arr, sizeof(int8_t) * 32);

    hipMemcpy(dev_sbox_arr, sbox, sizeof(uint8_t) * SBOX_LENGTH, hipMemcpyHostToDevice);
    hipMemcpy(dev_round_key, (aes_config_struct->round_key), sizeof(uint8_t) * aes_config_struct->round_key_length, hipMemcpyHostToDevice);
    hipMemcpy(dev_plain_text, (aes_config_struct->plain_text), sizeof(uint8_t) * aes_config_struct->plain_text_length, hipMemcpyHostToDevice);
    hipMemcpy(dev_comb_arr, comb_const, sizeof(int8_t) * 32, hipMemcpyHostToDevice);

    hipMemset(dev_cipher_text, 0, (sizeof(uint8_t) * aes_config_struct->plain_text_length));

    aes_ecb_gpu_encryption_kernel<<<block_count, THREADS_PER_BLOCK, smem_size>>>(dev_sbox_arr, dev_round_key, aes_config_struct->round_key_length, dev_plain_text, aes_config_struct->plain_text_length, num_rounds, dev_comb_arr, dev_cipher_text);

    hipDeviceSynchronize();

    hipMemcpy(aes_config_struct->cipher_text, dev_cipher_text, sizeof(uint8_t) * aes_config_struct->plain_text_length, hipMemcpyDeviceToHost);

#if DEBUG
    printf("\nPrinting encrypted IV...\n");
    for(int i = 0; i < aes_config_struct->plain_text_length; i++)
    {
        printf("0x%02x ", aes_config_struct->cipher_text[i]);
    }
#endif
    
    // XOR the encrypted IV with the plain text 
    for(int i = 0; i < aes_config_struct->plain_text_length; i++)
    {
        aes_config_struct->cipher_text[i] = aes_config_struct->cipher_text[i] ^ plain_text_buf[i];
    }

    delete [] ctr_buf;
    hipFree(dev_sbox_arr);
    hipFree(dev_round_key);
    hipFree(dev_plain_text);
    hipFree(dev_cipher_text);
    hipFree(dev_comb_arr);
}

// Function to encrypt the buffer in ECB mode
void aes_encrypt_ecb(aes_struct* aes_config_struct)
{
    // Calculate the number of blocks needed
    int block_count = (aes_config_struct->plain_text_length + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK;

    // Calculate the size required for the shared memory
    int smem_size = sizeof(uint8_t) * (SBOX_LENGTH + 2*THREADS_PER_BLOCK + AES256_ROUND_KEY_LENGTH) + sizeof(int8_t) * 32;

    uint8_t num_rounds;

    if(aes_config_struct->aes_key_length == AES128_KEY_SIZE*8)
    {
        num_rounds = 11;
    }
    else
    {
        num_rounds = 15;
    }

    // Buffer for device arrays
    uint8_t *dev_sbox_arr, *dev_round_key, *dev_plain_text, *dev_cipher_text;
    int8_t *dev_comb_arr;

    // Allocate memory to the device buffers
    hipMalloc((void**)&dev_sbox_arr, sizeof(uint8_t) * SBOX_LENGTH);
    hipMalloc((void**)&dev_round_key, sizeof(uint8_t) * aes_config_struct->round_key_length);
    hipMalloc((void**)&dev_plain_text, sizeof(uint8_t) * aes_config_struct->plain_text_length);
    hipMalloc((void**)&dev_cipher_text, sizeof(uint8_t) * aes_config_struct->plain_text_length);
    hipMalloc((void**)&dev_comb_arr, sizeof(int8_t) * 32);

    // Copy the values into device arrays
    hipMemcpy(dev_sbox_arr, sbox, sizeof(uint8_t) * SBOX_LENGTH, hipMemcpyHostToDevice);
    hipMemcpy(dev_round_key, (aes_config_struct->round_key), sizeof(uint8_t) * aes_config_struct->round_key_length, hipMemcpyHostToDevice);
    hipMemcpy(dev_plain_text, (aes_config_struct->plain_text), sizeof(uint8_t) * aes_config_struct->plain_text_length, hipMemcpyHostToDevice);
    hipMemcpy(dev_comb_arr, comb_const, sizeof(int8_t) * 32, hipMemcpyHostToDevice);

    // Set the cipher buffer to 0
    hipMemset(dev_cipher_text, 0, (sizeof(uint8_t) * aes_config_struct->plain_text_length));

    // Call the kernel function
    aes_ecb_gpu_encryption_kernel<<<block_count, THREADS_PER_BLOCK, smem_size>>>(dev_sbox_arr, dev_round_key, aes_config_struct->round_key_length, dev_plain_text, aes_config_struct->plain_text_length, num_rounds, dev_comb_arr, dev_cipher_text);

    hipDeviceSynchronize();

    // Copy the calculated cipher from the device array to host
    hipMemcpy(aes_config_struct->cipher_text, dev_cipher_text, sizeof(uint8_t) * aes_config_struct->plain_text_length, hipMemcpyDeviceToHost);
    
    hipFree(dev_sbox_arr);
    hipFree(dev_round_key);
    hipFree(dev_plain_text);
    hipFree(dev_cipher_text);
    hipFree(dev_comb_arr);
}

__device__ inline uint8_t aes_galoi_mult(uint8_t num, uint8_t mult)
{
    // Calculate the Galois product
    return (mult == 0x03) ? (((num & 0x80) ? (num << 1) ^ 0x1B : (num << 1)) ^ num) : ((mult == 0x02) ? ((num & 0x80) ? (num << 1) ^ 0x1B : (num << 1)) : num);
}

// Kernel Function
__global__ void aes_ecb_gpu_encryption_kernel(const uint8_t* sbox_arr, uint8_t* round_key_arr, uint8_t round_key_length, uint8_t* plain_text_arr, int plain_text_length, uint8_t num_rounds, int8_t* comb_arr, uint8_t* cipher_text_arr)
{
    // Dynamic shared memory allocation
    extern __shared__ uint8_t smem[];

    // Calculate the smem size
    const int smem_size = sizeof(uint8_t) * (SBOX_LENGTH + 2*THREADS_PER_BLOCK + AES256_ROUND_KEY_LENGTH) + sizeof(int8_t) * 32;

    // Calculate thread_count
    uint32_t thread_count = threadIdx.x + blockIdx.x * blockDim.x;

    // Calculate for easier computation
    uint32_t temp = thread_count * 2;

    uint8_t state_element, curr_round = 0;
    uint32_t index;

#if DEBUG
    if(threadIdx.x == 0)
    {
        printf("\nPrinting sbox values passed to device...\n");
        for(int i = 0; i < SBOX_LENGTH; i++)
        {
            printf("0x%02x ", sbox_arr[i]);
        }

        printf("\nPrinting round key values passed to device...\n");
        for(int i = 0; i < round_key_length; i++)
        {
            printf("0x%02x ", round_key_arr[i]);
        }

        printf("\nPrinting plain text values passed to device...\n");
        for(int i = 0; i < plain_text_length; i++)
        {
            printf("0x%02x ", plain_text_arr[i]);
        }
    }
#endif
    
    // Copy values from buffers to shared memory
    if(threadIdx.x < 128)
    {
        smem[threadIdx.x * 2] = sbox_arr[threadIdx.x * 2];
        smem[threadIdx.x * 2 + 1] = sbox_arr[threadIdx.x * 2 + 1];
    }
    else if((threadIdx.x >= 128) && (threadIdx.x < (248)) && ((threadIdx.x - 128) * 2 < round_key_length))
    {
        smem[threadIdx.x * 2] = round_key_arr[(threadIdx.x - 128) * 2];
        smem[threadIdx.x * 2 + 1] = round_key_arr[(threadIdx.x - 128) * 2 + 1];
    }
    else if((threadIdx.x >= 248) && (threadIdx.x < 256))
    {
        smem[1520 + (threadIdx.x - 248)*4] = comb_arr[(threadIdx.x - 248)*4];
        smem[1521 + (threadIdx.x - 248)*4] = comb_arr[(threadIdx.x - 248)*4 + 1];
        smem[1522 + (threadIdx.x - 248)*4] = comb_arr[(threadIdx.x - 248)*4 + 2];
        smem[1523 + (threadIdx.x - 248)*4] = comb_arr[(threadIdx.x - 248)*4 + 3];
    }
    else if((threadIdx.x >= 256) && ((threadIdx.x - 256) * 2 < plain_text_length))
    {
        smem[(threadIdx.x - 8) * 2] = plain_text_arr[(blockIdx.x * blockDim.x + (threadIdx.x - 256) * 2)];
        smem[(threadIdx.x - 8) * 2 + 1] = plain_text_arr[(blockIdx.x * blockDim.x + (threadIdx.x - 256) * 2) + 1];
    }

    // Allocate pointers in shared memory
    uint8_t* sbox = (uint8_t*)smem;
    uint8_t* round_key = (uint8_t*)&sbox[SBOX_LENGTH];
    uint8_t* plain_text = (uint8_t*)&smem[496];
    uint8_t* cipher_text = (uint8_t*)&smem[1008];
    int8_t* shift_row_const = (int8_t*)&smem[1520];
    uint8_t* mult_arr = (uint8_t*)&smem[1536];

    // Wait till all threads complete transfer to shared mem
    __syncthreads();

#if DEBUG
    if(thread_count == 0)
    {
        printf("\nSmem size = %d\n", smem_size);

        printf("\nPrinting sbox values in shared memory...\n");
        for(int i = 0; i < SBOX_LENGTH; i++)
        {
            printf("0x%02x ", sbox[i]);
        }

        printf("\nPrinting round key values in shared memory...\n");
        for(int i = 0; i < round_key_length; i++)
        {
            printf("0x%02x ", round_key[i]);
        }

        printf("\nPrinting plain text values in shared memory...\n");
        for(int i = 0; i < THREADS_PER_BLOCK; i++)
        {
            printf("%02x", plain_text[i]);
        }

        printf("\nPrinting shift row const values in shared memory...\n");
        for(int i = 0; i < 16; i++)
        {
            printf("%d ", shift_row_const[i]);
        }
        
        printf("\nPrinting mult arr values in shared memory...\n");
        for(int i = 0; i < 16; i++)
        {
            printf("0x%02x ", mult_arr[i]);
        }
    }
#endif

    if(thread_count < plain_text_length)
    {
        // Ensure that the threads are within bounds
        index = thread_count;
        temp = thread_count % 16;

        // Add round key
        state_element = plain_text[threadIdx.x] ^ round_key[temp];
        ++curr_round;

        for(; curr_round < num_rounds - 1; curr_round++)
        {
            // Substitute S-box matrix
            state_element = sbox[state_element];

            // Shift rows. Change index values and move the elements
            index = (index + shift_row_const[temp]) % 512;
            cipher_text[index] = state_element;

            // Wait for all threads to complete
            __syncwarp();
            
            index = threadIdx.x;

            // Mix Columns step 
            state_element = aes_galoi_mult(cipher_text[(index / 4) * 4], mult_arr[(index % 4) * 4]) ^ aes_galoi_mult(cipher_text[(index / 4) * 4 + 1], mult_arr[(index % 4) * 4 + 1]) ^ aes_galoi_mult(cipher_text[(index / 4) * 4 + 2], mult_arr[(index % 4) * 4 + 2]) ^ aes_galoi_mult(cipher_text[(index / 4) * 4 + 3], mult_arr[(index % 4) * 4 + 3]);
            
            // Add round key step
            state_element = state_element ^ round_key[curr_round*AES_BLK_LENGTH + (temp)];

            // Copy the state element back to shared memory buffer
            cipher_text[index] = state_element;
        }

        // Substitution step
        state_element = sbox[state_element];

        index = threadIdx.x;

        // Shift row step. Change index values instead of moving the elements
        index = (index + shift_row_const[temp]) % 512;

        // Add round key
        state_element = state_element ^ round_key[(curr_round)*AES_BLK_LENGTH + (temp) + shift_row_const[temp]];
        
        // Copy element back to the shared memory buffer
        cipher_text[index] = state_element;

        // Finally copy from shared memory to the device buffer
        cipher_text_arr[thread_count] = cipher_text[threadIdx.x];
    }
}
